#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "bfs.h"
#include "sparse.h"
#include "labirint_io.h"

void check_input(LabIOMatrix const & mat, int start_row, int start_col,
                 int stop_row, int stop_col);

///////// Vaša CUDA jezgra dolazi ovdje ////////////////////

__global__
void bfs_kernel(CSRMat *incidence, int *level, int *newVertVisited, int currentLevel) {
    int vertex = blockIdx.x * blockDim.x + threadIdx.x;
    if (vertex < incidence->nrows) {
        if (level[vertex] == currentLevel - 1) {
            for (int edge = incidence->rowPtrs[vertex]; edge < incidence->rowPtrs[vertex + 1]; ++edge) {
                int neighbor = incidence->colIdx[edge];
                if (level[neighbor] == -1) {
                    level[neighbor] = currentLevel;
                    *newVertVisited = 1;
                }
            }
        }
    }
}
////////////////////////////////////////////////////////////

int main(int argc, char * argv[])
{
    int start_row = -1; // polazna točka row
    int start_col = -1; // polazna točka col
    int stop_row = -1;  // završna točka row
    int stop_col = -1;  // završna točka col
    std::string file_name = "labirint.txt"; // ulazna datoteka s labirintom

    if(argc >= 6){
        start_row = std::stoi(argv[1]);
        start_col = std::stoi(argv[2]);
        stop_row = std::stoi(argv[3]);
        stop_col = std::stoi(argv[4]);
        file_name = argv[5];
    }
    else{
        std::cerr << "Upotreba: " << argv[0] << " start_row start_col stop_row stop_col file_name\n";
        std::cerr << "Brojevi stupaca i redaka idu od nule.\n";
        std::exit(1);
    }

    // Kreiraj labirint. Labirint je zadan s matricom tipa LabMatrix.
    LabIOMatrix mat;
    mat.read(file_name);
    check_input(mat, start_row, start_col, stop_row, stop_col);

    // Kreiraj graf iz labirinta. Funkcija vraća matricu incidencije koja je ovdje dana kao
    // puna matrica.
    IncidenceMat incidence(mat);
    CSRMat csr_incidence(incidence);
    CSCMat csc_incidence(incidence);

    // csr_incidence.print();
    // csc_incidence.print();

    int start_idx = mat(start_row, start_col);
    int stop_idx  = mat(stop_row,stop_col);
    std::cout << "start index = " << start_idx << ", stop index = " << stop_idx << "\n";

    /// VAŠ CUDA kod  DOLAZI OVDJE /////////////////////////////////////////
    // ALOCIRAJ MEMORIJU NA GPU, KOPIRAJ PODATKE S CPU NA GPU,
    // POZOVI JEZGRU, KOPIRAJ LEVEL POLJE S GPU NA CPU.

    CSRMat *d_csr_incidence;
    hipMalloc((void**) (&d_csr_incidence), sizeof(CSRMat));
    hipMemcpy(d_csr_incidence, &csr_incidence, sizeof(CSRMat), hipMemcpyHostToDevice);

    int *d_rowPtrs, *d_colIdx;
    hipMalloc(&d_rowPtrs, (csr_incidence.nrows + 1) * sizeof(int));
    hipMalloc(&d_colIdx, csr_incidence.nelem * sizeof(int));

    hipMemcpy(d_rowPtrs, csr_incidence.rowPtrs, (csr_incidence.nrows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colIdx, csr_incidence.colIdx, (csr_incidence.nelem) * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(&(d_csr_incidence->rowPtrs), &d_rowPtrs, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_csr_incidence->colIdx), &d_colIdx, sizeof(int*), hipMemcpyHostToDevice);

    std::vector<int> level(csr_incidence.nrows, -1);
    level[start_idx] = 0;

    int *d_level;
    hipMalloc(&d_level, level.size() * sizeof(int));
    hipMemcpy(d_level, level.data(), level.size() * sizeof(int), hipMemcpyHostToDevice);

    int found_new = 0;
    int *d_found_new;
    hipMalloc(&d_found_new, sizeof(int));

    int current_level = 1;

    const int BLOCK = 128;
    const int GRID = (mat.no_blocks() + BLOCK - 1) / BLOCK;
    do {
        found_new = 0;
        hipMemcpy(d_found_new, &found_new, sizeof(int), hipMemcpyHostToDevice);
        bfs_kernel<<<BLOCK, GRID>>>(d_csr_incidence, d_level, d_found_new, current_level++);
        hipDeviceSynchronize();
        hipMemcpy(&found_new, d_found_new, sizeof(int), hipMemcpyDeviceToHost);
    } while (found_new);


    hipMemcpy(level.data(), d_level, level.size() * sizeof(int), hipMemcpyDeviceToHost);

    ///////////////////////////////////////////////////////////////////////


    std::vector<int>  path;  // STAZA
    // IZRAČUNAJ STAZU
    find_path(csc_incidence, stop_idx, level, path);
    // PRINTAJ STAZU U DATOTEKU
    mat.print_ascii("out_"+base_name(file_name), path);

    // POČISTITE MEMORIJU ////////////////////
    hipFree(d_found_new);
    hipFree(d_level);
    hipFree(d_rowPtrs);
    hipFree(d_colIdx);
    hipFree(d_csr_incidence);
    ///////////////////////////////////////////

    return 0;
}
